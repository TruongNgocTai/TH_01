
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char * fileName, int &numChannels, int &width, int &height, uint8_t * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	if (strcmp(type, "P2") == 0)
		numChannels = 1;
	else if (strcmp(type, "P3") == 0)
		numChannels = 3;
	else 
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); // In this exercise, we don't touch other types
		exit(EXIT_FAILURE); 
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);
	
	uint8_t max_val;
	fscanf(f, "%hhu", &max_val);
	if (max_val > 255)
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); // In this exercise, we assume 1 byte per value
		exit(EXIT_FAILURE); 
	}

	pixels = (uint8_t *)malloc(width * height * numChannels);
	for (int i = 0; i < width * height * numChannels; i++)
		fscanf(f, "%hhu", &pixels[i]);

	fclose(f);
}

void writePnm(char * fileName, int numChannels, int width, int height, uint8_t * pixels)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	
	
	if (numChannels == 1)
		fprintf(f, "P2\n");
	else if (numChannels == 3)
		fprintf(f, "P3\n");
	else
	{
		fclose(f);
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	fprintf(f, "%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height * numChannels; i++)
		fprintf(f, "%hhu\n", pixels[i]);
	
	fclose(f);
}

void compare2Pnms(char * fileName1, char * fileName2)
{
	int numChannels1, width1, height1;
	uint8_t * pixels1;
	readPnm(fileName1, numChannels1, width1, height1, pixels1);

	int numChannels2, width2, height2;
	uint8_t * pixels2;
	readPnm(fileName2, numChannels2, width2, height2, pixels2);

	if (numChannels1 != numChannels2)
	{
		printf("'%s' is DIFFERENT from '%s' (num channels: %i vs %i)\n", fileName1, fileName2, numChannels1, numChannels2);
		return;
	}
	if (width1 != width2)
	{
		printf("'%s' is DIFFERENT from '%s' (width: %i vs %i)\n", fileName1, fileName2, width1, width2);
		return;
	}
	if (height1 != height2)
	{
		printf("'%s' is DIFFERENT from '%s' (width: %i vs %i)\n", fileName1, fileName2, height1, height2);
		return;
	}
	float mae = 0;
	for (int i = 0; i < width1 * height1 * numChannels1; i++)
	{
		mae += abs((int)pixels1[i]-(int)pixels2[i]);
	}
	mae /= (width1 * height1 * numChannels1);
	printf("The average pixel difference between '%s' and '%s': %f\n", fileName1, fileName2, mae);
}

void convertRgb2GrayByHost(uint8_t * inPixels, uint8_t * outPixels, int width, int height)
{
	// TODO
	int size = width * height;

	for(int i = 0; i < size; i++){
		outPixels[i] = 0.299 * inPixels[i*3] +
						 0.114 * inPixels[i*3 + 2] + 
						 0.587 * inPixels[i*3 + 1];
	}
}

__global__ void convertRgb2GrayByDevice(uint8_t * inPixels, uint8_t * outPixels, int width, int height)
{
	// TODO
	int i_r = blockIdx.y * blockDim.y + threadIdx.y;
	int i_c = blockIdx.x * blockDim.x + threadIdx.x;

	if(i_c < width && i_r < height){
		outPixels[i_r * width + i_c] = 0.299 * inPixels[(i_r * width + i_c)*3] + 
										0.114 * inPixels[(i_r * width + i_c)*3 + 2] + 
										0.587 * inPixels[(i_r * width + i_c)*3 + 1];
	}
}


int main(int argc, char ** argv)
{
	// -----READ INPUT DATA-----
	if (argc < 5 || argc > 7)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}

	int numChannels, width, height;
	uint8_t * inPixels;
	readPnm(argv[1], numChannels, width, height, inPixels);
	printf("Image size (width x height): %i x %i\n", width, height);
	// -----PROCESS INPUT DATA-----
	uint8_t * outPixels= (uint8_t *)malloc(width * height);
	GpuTimer timer;
    timer.Start();
	if (strcmp(argv[4], "cpu") == 0){ // Use CPU
		convertRgb2GrayByHost(inPixels, outPixels, width, height);
	}
	else // Use GPU
	{
		// TODO: Query and print GPU name and compute capability
		hipDeviceProp_t prop;
		printf("GPU name: %s\n", prop.name);
		printf("GPU compute capability: %d\n", prop.major);
		printf("GPU compute capability: %d\n", prop.minor);

		// TODO: Allocate device memories
		uint8_t *d_inPixels, *d_outPixels;
		CHECK(hipMalloc(&d_inPixels, width * height * numChannels));
		CHECK(hipMalloc(&d_outPixels, width * height));

		// TODO: Copy data to device memories
		CHECK(hipMemcpy(d_inPixels, inPixels, width * height * numChannels, hipMemcpyHostToDevice));

		// TODO: Set block size (already done for you) and grid size,
		//       and invoke kernel function with these settings (remember to check kernel error)
		dim3 blockSize(32, 32); // Default
		if (argc == 7)
		{
			blockSize.x = atoi(argv[5]);
			blockSize.y = atoi(argv[6]);
		}
		dim3 gridSize(16, 16);
		convertRgb2GrayByDevice<<<gridSize, blockSize>>>(d_inPixels, d_outPixels, width, height);

		// TODO: Copy result from device memories
		CHECK(hipMemcpy(outPixels, d_outPixels, width * height, hipMemcpyDeviceToHost));

		// TODO: Free device memories
		hipFree(d_inPixels);
		hipFree(d_outPixels);
	}
	timer.Stop();
    float time = timer.Elapsed();
    printf("Processing time: %f ms\n", time);

    // -----WRITE OUTPUT DATA TO FILE-----
	writePnm(argv[2], 1, width, height, outPixels);

	free(inPixels);
	free(outPixels);

	// -----CHECK CORRECTNESS-----
	compare2Pnms(argv[2], argv[3]);
}